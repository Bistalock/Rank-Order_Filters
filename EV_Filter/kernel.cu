#include "hip/hip_runtime.h"
/*
 *      Project Name: Rank-Order EV filter
 *      Date Written: October 17, 2014
 *      Created by: Gabriel Del Pino. Portions inspired by Hiroyuki Plumlee.
 */  

#include "hip/hip_runtime.h"
#include <cmath>



extern "C"
{
	// **TODO: incorporate other transformation methods - currently only works with the mean values
	// Function requires a 1 dimentional buffer of an image, along as various parameters
	// The image array buffer is later converted to a 3 dimentional dynamic array in the format of inputImage[Samples][Height][Width]
	__declspec(dllexport) unsigned char* getImage(unsigned char* mirrorImageBuffer, int height, int width, int samples, int kernelHeight, int kernelWidth, int EV)
	{
		int offsetHeight = (kernelHeight - 3) / 2 + 1; // calculation of the center 
		int offsetWidth = (kernelWidth - 3) / 2 + 1; // calculation of the center  
		
		// Mirrorimage before the creation of the kernel
		int MirroredHeight = height + (offsetHeight * 2);
		int MirroredWidth = width + (offsetWidth * 2);
		
		// Initiallizing the 3 dimentional dynamic array as a series of pointers
		unsigned char*** mirrorImage = new unsigned char**[samples];
		
		// Converting the 1 dimentional array to a 3 dimentional array for C++
		for (int k = 0; k < samples; k++)
		{
			mirrorImage[k] = new unsigned char*[MirroredHeight];
			for (int i = 0; i < MirroredHeight; i++)
			{
				mirrorImage[k][i] = new unsigned char[MirroredWidth];
				for (int j = 0; j < MirroredWidth; j++)
				{            
					mirrorImage[k][i][j] = mirrorImageBuffer[((MirroredWidth * samples) * i) + (samples * j) + k];
				}
			}
		}

		
		// initiallizing the kernel widow
		unsigned char** window = new unsigned char*[kernelHeight]; // the kernel window for each sample
		// initiallizing the array values to zero
		for (int i = 0; i < kernelHeight; i++)
		{
			window[i] = new unsigned char[kernelWidth];
			for (int j = 0; j < kernelWidth; j++)
				{
					window[i][j] = 0;
				}
		}
		
		// initializing the resulting image
		unsigned char*** processedImage = new unsigned char**[samples, height, width]; // the resulting image after the filtering
		// initiallizing the array values to zero
		for (int k = 0; k < samples; k++)
		{
			processedImage[k] = new unsigned char*[height];
			for (int i = 0; i < height; i++)
			{
				processedImage[k][i] = new unsigned char[width];
				for (int j = 0; j < width; j++)
				{
					processedImage[k][i][j] = 0;
				}
			}
		}
		
		//  Perfect CUDA zone
		#pragma region Applying kernel to every pixel in image
		// Consider adding if statement for samples
		for (int k = 0; k < samples; k++)
		{ 
			for (int i = offsetHeight; i < MirroredHeight - offsetHeight; i++)
			{
				for (int j = offsetWidth; j < MirroredWidth - offsetWidth; j++)
				{
					
					// Begin creation of kernel
					double addedsum = 0;
					double selected = 0;
					
					int mean; // initializing the mean variable
					
					// the current pixel that is being processed. Also the center pixel of the kernel to be created.
					unsigned char temp = mirrorImage[k][i][j]; 
					
					// *** this loop is commented because we may create the kernel at the same time we calculate a tranformation method (mean, median) *** // 
					//for (int l = 0; l < kernelHeight; l++) // kernel height
					//{
					//    for (int m = 0; m < kernelWidth; m++) // kernel width
					//    {
					//        window[l, m] = mirrorImage[k][i - offsetHeight + l][j - offsetWidth + m]; // applying the intensity values to the kernel window
					//	  }
					//}
					
					// loops collecting the pixels within the kernel meeting the EV criterion and adding to addedsum temp var
					for (int l = 0; l < kernelHeight; l++) // kernel height
					{
						for (int m = 0; m < kernelWidth; m++) // kernel width
						{
							window[l][m] = mirrorImage[k][i - offsetHeight + l][j - offsetWidth + m]; // applying the intensity values to the kernel window
							/* this if statement is commented because of the fact that Project VEGA seems to take the 
							 * center pixel into account as well... */
							//if (l != offsetHeight || m != offsetWidth) // as long as the pixel to be processed in not the center pixel
							//{
							    // ***** obtaning the absolute difference of the pixel within the kernel to the offset ***** //
								if (abs(window[l][m] - temp) <= EV) 
								{
									// **TODO: maybe add switch/case statement for mean and median methods?
									
									// used for mean method:
									selected++; // incrementing the counter for the pixel within the kernel
									addedsum += window[l][m]; // adding the intenities within the kernel to one variable
								}
							//}
						}
					}
					
					// calulating the mean of kernel                     
					// if no pixels in kernel matched EV parameter, set to the center pixel. Else, add all values and divided by the number of elements
					selected == 0 ? mean = window[offsetHeight][offsetWidth] : mean = int(addedsum / selected); 
					
					//if (selected == 0)
					//{
					// mean = window[offsetHeight][offsetWidth]; // the center coordinate of a kernel
					//}
					//else
					//{
					// mean = int(addedsum / selected); // all added values, divided by the number of elements
					//}
					
					// tranfering the mean values to the resulting image
					processedImage[k][i - offsetHeight][j - offsetWidth] = unsigned char(mean);                      
				}
			}
		} // end of EV filtering
		#pragma endregion
		
		// Initializing the 1 dimentional array buffer
		unsigned char* processedImageBuffer = new unsigned char[samples * width * height];
		
		// converting the 3 dimentional array to a 1 dimentional buffer for C#
		for (int k = 0; k < samples; k++) // the samples per pixel
		{
			for (int i = 0; i < height; i++) // the height (or column) of an image
			{
				for (int j = 0; j < width; j++) // the width (or row) of an image
				{
					processedImageBuffer[((width * samples) * i) + (samples * j) + k] = processedImage[k][i][j]; // assigining all values
				}
			}
		}
		
		// outputs the 1 dimentional array buffer.
		return processedImageBuffer;
	}
}
